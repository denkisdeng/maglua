#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "memory.hpp"
#include <stdio.h>

#define KCHECK \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  __FILE__, __LINE__-1, hipGetErrorString(i));\
}

#define KCHECK_FL(f,l) \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  f, l, hipGetErrorString(i));\
}


#define SEGFAULT \
{ \
	long* i = 0; \
	*i = 5; \
}

static size_t memTotal()
{
	size_t free, total;
	//hipMemGetInfo(&free, &total);
	hipMemGetInfo(&free, &total);       

	return total;
}

static size_t memLeft()
{
// 	hipError_t res;
	size_t free, total;
	//hipMemGetInfo(&free, &total);
	hipMemGetInfo(&free, &total); 

	return free;
}

#define CHECKCALL_FL(expression, file, line) \
{ \
	const hipError_t err = (expression); \
	if(err != hipSuccess) \
	{ \
		printf("(%s:%i) %s => (%i)%s\n", file, line, #expression, err, hipGetErrorString(err)); \
		fprintf(logfile,"(%s:%i) %s => (%i)%s\n", file, line, #expression, err, hipGetErrorString(err)); \
	} \
}

#define CHECKCALL_FLe(lval,expression, file, line)	\
{ \
	lval = (expression); \
	if(lval != hipSuccess) \
	{ \
		printf("(%s:%i) %s => (%i)%s\n", file, line, #expression, lval, hipGetErrorString(lval)); \
		fprintf(logfile,"(%s:%i) %s => (%i)%s\n", file, line, #expression, lval, hipGetErrorString(lval)); \
	} \
}

#define CHECKCALL(expression)  CHECKCALL_FL(expression, __FILE__, __LINE__)
#define CHECKCALLe(lval,expression)  CHECKCALL_FLe(lval,expression, __FILE__, __LINE__)

static FILE* logfile = 0;
hipError_t malloc_device_(void** d_v, size_t n, const char* file, unsigned int line)
{
    if(!logfile)
    {
	logfile = fopen("malloc.log", "w");
    }

    hipError_t err;
// 	printf("malloc_device %i bytes\n", n);
    CHECKCALL_FLe(err,hipMalloc(d_v, n), file, line);

	
	fprintf(logfile, "[%10lu/%10lu] (%s:%i) %8li %p\n", memTotal()-memLeft(), memTotal(), file, line, n, *d_v);
// 	fprintf(logfile, "free %i bytes\n",  memLeft());
	fflush(logfile);

	//TODO here we check for fail and compress if needed
	
	return err; //eventually this will reflect succesfulness of malloc
}

void free_device_(void* d_v, const char* file, unsigned int line)
{
	CHECKCALL_FL(hipFree(d_v), file,line);
}

hipError_t malloc_host_(void** h_v, size_t n, const char* file, unsigned int line)
{
    hipError_t err;
    CHECKCALL_FLe(err,hipHostMalloc(h_v, n),file,line);
    return err; //to mirror malloc_device
}

void free_host_(void* h_v, const char* file, unsigned int line)
{
	CHECKCALL_FL(hipHostFree(h_v),file,line);
}





void memcpy_d2d_(void* d_dest, void* d_src, size_t n, const char* file, const unsigned int line)
{
    CHECKCALL_FL(hipMemcpy(d_dest, d_src, n, hipMemcpyDeviceToDevice),file,line);
}

void memcpy_d2h_(void* h_dest, void* d_src, size_t n, const char* file, const unsigned int line)
{
    CHECKCALL_FL(hipMemcpy(h_dest, d_src, n, hipMemcpyDeviceToHost),file,line);
}

void memcpy_h2d_(void* d_dest, void* h_src, size_t n, const char* file, const unsigned int line)
{
    CHECKCALL_FL(hipMemcpy(d_dest, h_src, n, hipMemcpyHostToDevice),file,line);
}
