#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "memory.hpp"
#include <stdio.h>

#define KCHECK \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  __FILE__, __LINE__-1, hipGetErrorString(i));\
}

#define KCHECK_FL(f,l) \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  f, l, hipGetErrorString(i));\
}


#define SEGFAULT \
{ \
	long* i = 0; \
	*i = 5; \
}

static size_t memTotal()
{
	size_t free, total;
	//hipMemGetInfo(&free, &total);
	hipMemGetInfo(&free, &total);       

	return total;
}

static size_t memLeft()
{
// 	hipError_t res;
	size_t free, total;
	//hipMemGetInfo(&free, &total);
	hipMemGetInfo(&free, &total); 

	return free;
}

#define CHECKCALL_FL(expression, file, line) \
{ \
	const hipError_t err = (expression); \
	if(err != hipSuccess) \
	{ \
		printf("(%s:%i) %s => (%i)%s\n", file, line, #expression, err, hipGetErrorString(err)); \
		fprintf(logfile,"(%s:%i) %s => (%i)%s\n", file, line, #expression, err, hipGetErrorString(err)); \
	} \
}

#define CHECKCALL_FLe(lval,expression, file, line)	\
{ \
	lval = (expression); \
	if(lval != hipSuccess) \
	{ \
		printf("(%s:%i) %s => (%i)%s\n", file, line, #expression, lval, hipGetErrorString(lval)); \
		fprintf(logfile,"(%s:%i) %s => (%i)%s\n", file, line, #expression, lval, hipGetErrorString(lval)); \
	} \
}

#define CHECKCALL(expression)  CHECKCALL_FL(expression, __FILE__, __LINE__)
#define CHECKCALLe(lval,expression)  CHECKCALL_FLe(lval,expression, __FILE__, __LINE__)

static FILE* logfile = 0;
hipError_t malloc_device_(void** d_v, size_t n, const char* file, unsigned int line)
{
    if(!logfile)
    {
	logfile = fopen("malloc.log", "w");
    }

    hipError_t err;
// 	printf("malloc_device %i bytes\n", n);
    CHECKCALL_FLe(err,hipMalloc(d_v, n), file, line);

	
	fprintf(logfile, "[%10lu/%10lu] (%s:%i) %8li %p\n", memTotal()-memLeft(), memTotal(), file, line, n, *d_v);
// 	fprintf(logfile, "free %i bytes\n",  memLeft());
	fflush(logfile);

	//TODO here we check for fail and compress if needed
	
	return err; //eventually this will reflect succesfulness of malloc
}

void free_device_(void* d_v, const char* file, unsigned int line)
{
	CHECKCALL_FL(hipFree(d_v), file,line);
}

hipError_t malloc_host_(void** h_v, size_t n, const char* file, unsigned int line)
{
    hipError_t err;
    CHECKCALL_FLe(err,hipHostMalloc(h_v, n),file,line);
	if(!logfile)
    {
		logfile = fopen("malloc.log", "w");
    }
	fprintf(logfile, "malloc_host %i bytes from %s:%i\n", (int)n, file, line);
	fflush(logfile);
	return err; //to mirror malloc_device
}

void free_host_(void* h_v, const char* file, unsigned int line)
{
	CHECKCALL_FL(hipHostFree(h_v),file,line);
}



hipError_t ARRAYCUDA_API malloc_dh_(void** d, void** h, size_t n, const char* file, const unsigned int line)
{
	        malloc_host_(h, n, file, line);
	return  malloc_device_(d, n, file, line);
}


void ARRAYCUDA_API free_dh_(void* d, void* h, const char* file, unsigned int line)
{
	free_device_(d, file, line);
	free_host_(h, file, line);
}



void memcpy_d2d_(void* d_dest, void* d_src, size_t n, const char* file, const unsigned int line)
{
    CHECKCALL_FL(hipMemcpy(d_dest, d_src, n, hipMemcpyDeviceToDevice),file,line);
}

void memcpy_d2h_(void* h_dest, void* d_src, size_t n, const char* file, const unsigned int line)
{
    CHECKCALL_FL(hipMemcpy(h_dest, d_src, n, hipMemcpyDeviceToHost),file,line);
}

void memcpy_h2d_(void* d_dest, void* h_src, size_t n, const char* file, const unsigned int line)
{
    CHECKCALL_FL(hipMemcpy(d_dest, h_src, n, hipMemcpyHostToDevice),file,line);
}

















typedef struct work_space_device_memory
{
	int refcount;
	void* d_memory[5];
	size_t size[5];
} work_space_device_memory;

static work_space_device_memory WS_MEM = {0};

ARRAYCUDA_API void  registerWS()
{
	if(WS_MEM.refcount == 0) //initialize
	{
		for(int i=0; i<5; i++)
		{
			WS_MEM.d_memory[i] = 0;
			WS_MEM.size[i] = 0;
		}
	}
	
	WS_MEM.refcount++;
}

ARRAYCUDA_API void  unregisterWS()
{
	WS_MEM.refcount--;
	if(WS_MEM.refcount == 0)
	{
		for(int i=0; i<5; i++)
		{
			if(WS_MEM.d_memory[i])
				free_device(WS_MEM.d_memory[i]);
			WS_MEM.d_memory[i] = 0;
			WS_MEM.size[i] = 0;
		}
	}
}

ARRAYCUDA_API void  getWSMem5_(void** ptr1,   size_t size1, 
			   void** ptr2, size_t size2, 
			   void** ptr3, size_t size3,
			   void** ptr4, size_t size4,
			   void** ptr5, size_t size5)
{
	void**  ptr[5] = {ptr1, ptr2, ptr3, ptr4, ptr5};
	size_t size[5] = {size1, size2, size3, size4, size5};
	
	for(int i=0; i<5; i++)
	{
		if(size[i] > WS_MEM.size[i])
		{
			if(WS_MEM.d_memory[i])
				free_device(WS_MEM.d_memory[i]);
			malloc_device(&(WS_MEM.d_memory[i]), size[i]);
			WS_MEM.size[i] = size[i];
		}
		if(ptr[i])
			*ptr[i] = WS_MEM.d_memory[i];
	}
}


ARRAYCUDA_API void  getWSMem4_(
			   void** ptr1, size_t size1, 
			   void** ptr2, size_t size2, 
			   void** ptr3, size_t size3,
			   void** ptr4, size_t size4)
{
	getWSMem5(ptr1, size1, ptr2, size2, ptr3, size3, ptr4, size4, 0, 0); 
}

ARRAYCUDA_API void  getWSMem3_(
			   void** ptr1, size_t size1, 
			   void** ptr2, size_t size2, 
			   void** ptr3, size_t size3)
{
	getWSMem4(ptr1, size1, ptr2, size2, ptr3, size3, 0, 0);
}
ARRAYCUDA_API void  getWSMem2_(
			   void** ptr1, size_t size1, 
			   void** ptr2, size_t size2)
{
	getWSMem3(ptr1, size1, ptr2, size2, 0, 0);
}

ARRAYCUDA_API void  getWSMem1_(
			   void** ptr1, size_t size1)
{
	getWSMem2(ptr1, size1, 0, 0);
}





