#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <stdio.h>

#include "spinoperationexchange.hpp"

#define KCHECK \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  __FILE__, __LINE__-1, hipGetErrorString(i));\
}


__global__ void do_exchange(
	const double* d_sx, const double* d_sy, const double* d_sz,
	const double* d_strength, const int* d_neighbour, const int max_neighbours,
	double* d_hx, double* d_hy, double* d_hz,
	const int nxyz
	)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= nxyz) return;
	
	d_hx[i] = 0;
	d_hy[i] = 0;
	d_hz[i] = 0;
	// not all sites have max_neighbours but we've dummied the
	// fromsite and zero'd the strength so it doesn't matter
	for(int j=0; j<max_neighbours; j++)
	{
		const int p = i * max_neighbours + j;
		const int k = d_neighbour[p];
		const double strength = d_strength[p];

		d_hx[i] += strength * d_sx[k];
		d_hy[i] += strength * d_sy[k];
		d_hz[i] += strength * d_sz[k];
	}
}


void cuda_exchange(
	const double* d_sx, const double* d_sy, const double* d_sz,
	const double* d_strength, const int* d_neighbour, const int max_neighbours,
	double* d_hx, double* d_hy, double* d_hz,
	const int nx, const int ny, const int nz
					)
{
	const int nxyz = nx*ny*nz;
	const int threads = 128;
	const int blocks = nxyz / threads + 1;
	
	do_exchange<<<blocks, threads>>>(
			d_sx, d_sy, d_sz,
			d_strength, d_neighbour, max_neighbours,
			d_hx, d_hy, d_hz, 
			nxyz);

	KCHECK;
}



__global__ void do_exchange_compressed(
	const double* d_sx, const double* d_sy, const double* d_sz,
	const ex_compressed_struct* d_LUT, const unsigned char* d_idx, const int max_neighbours,
	double* d_hx, double* d_hy, double* d_hz,
	const int nxyz)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= nxyz) return;
	
	const ex_compressed_struct* e = & d_LUT[ (int)d_idx[i] * max_neighbours ];
	
	d_hx[i] = 0;
	d_hy[i] = 0;
	d_hz[i] = 0;
	for(int j=0; j<max_neighbours; j++)
	{
		const int p = (i + e[j].offset) % nxyz;
		const double strength = e[j].strength;

		d_hx[i] += strength * d_sx[p];
		d_hy[i] += strength * d_sy[p];
		d_hz[i] += strength * d_sz[p];
	}
}


void cuda_exchange_compressed(
	const double* d_sx, const double* d_sy, const double* d_sz,
	const ex_compressed_struct* d_LUT, const unsigned char* d_idx, const int max_neighbours,
	double* d_hx, double* d_hy, double* d_hz, 
	const int nxyz)
{
	const int threads = 256;
	const int blocks = nxyz / threads + 1;

	do_exchange_compressed<<<blocks, threads>>>(
		d_sx, d_sy, d_sz,
		d_LUT, d_idx, max_neighbours,
		d_hx, d_hy, d_hz, 
		nxyz);
	KCHECK;
}
