#include <hip/hip_runtime.h>

#include <math_functions.h>
#include <stdio.h>

// 
// A lot of the routines here have both alpha and d_alpha
// passed in. This is for local vs global values. Ditto
// for gamma
// 


#define CROSS(v, a, b) \
	v.x = a.y * b.z - a.z * b.y; \
	v.y = a.z * b.x - a.x * b.z; \
	v.z = a.x * b.y - a.y * b.x;
	
#if __CUDA_ARCH__ >= 200
#define FAST_DIV(x,y) __ddiv_rn(x,y)
#else
#define FAST_DIV(x,y) ((x)/(y))
#endif

#define CHECK \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  __FILE__, __LINE__-1, hipGetErrorString(i));\
}


__device__ double4 qconjugate(const double4 q)
{
	double4 r;
	r.x = -1.0 * q.x;
	r.y = -1.0 * q.y;
	r.z = -1.0 * q.z;
	r.w =        q.w;
	return r;
}

__device__ double4 qmult(const double4 a, const double4 b)
{
	double4 ab;

	ab.w = a.w*b.w - a.x*b.x - a.y*b.y - a.z*b.z;
	ab.x = a.w*b.x + b.w*a.x  + a.y*b.z - a.z*b.y;
	ab.y = a.w*b.y + b.w*a.y  + a.z*b.x - a.x*b.z;
	ab.z = a.w*b.z + b.w*a.z  + a.x*b.y - a.y*b.x;

	return ab;
}

// quat mult without calculating the W component
__device__ double4 qmultXYZ(const double4 a, const double4 b)
{
	double4 ab;

	ab.x = a.w*b.x + b.w*a.x  + a.y*b.z - a.z*b.y;
	ab.y = a.w*b.y + b.w*a.y  + a.z*b.x - a.x*b.z;
	ab.z = a.w*b.z + b.w*a.z  + a.x*b.y - a.y*b.x;
	ab.w = 0;

	return ab;
}

// _1 will compute rhs and store in ws
// dS    -g           a
// -- = ---- S X (H +---S X H)
// dt   1+aa         |S|


template<int thermalOnlyFirstTerm>
__global__ void llg_quat_apply_1(
//	const int nx, const int ny, const int offset,
	const int nxyz,
	double alpha, const double* d_alpha,
	double* sx, double* sy, double* sz, double* sms,
	double* hx, double* hy, double* hz,
    double* htx, double* hty, double* htz,              // dm/dt thermal fields
	// ws1, ws2, ws3,     ws4
	double* wx, double* wy, double* wz, double* ww) 
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i >= nxyz)
		return;

	double hX = hx[i];
	double hY = hy[i];
	double hZ = hz[i];
	
	// subtracting thermal term from Heff in damping contribution
	if(thermalOnlyFirstTerm == 1 && htx)
	{
		hX -= htx[i];
		hY -= hty[i];
		hZ -= htz[i];
	}
	
	
	wx[i] = sy[i]*hZ - sz[i]*hY;
	wy[i] = sz[i]*hX - sx[i]*hZ;
	wz[i] = sx[i]*hY - sy[i]*hX;
	
	ww[i] = sms[i];
	if(ww[i] != 0)
	{
		if(d_alpha)
		{
			ww[i] = FAST_DIV(d_alpha[i], ww[i]);
		}
		else
		{
			ww[i] = FAST_DIV(alpha, ww[i]);
		}
	}
	wx[i] *= ww[i];
	wy[i] *= ww[i];
	wz[i] *= ww[i];
	
	wx[i] += hx[i];
	wy[i] += hy[i];
	wz[i] += hz[i];

	ww[i] = sqrt(wx[i]*wx[i] + wy[i]*wy[i] + wz[i]*wz[i]);
	
	//(wx, wy, wz) = (a / |S|) S x H
	// ww = | (wx, wy, wz) |
}


// _2 will compute the rest
// dS    -g           a
// -- = ---- S X (H +---S X H)
// dt   1+aa         |S|
// the rhs vec and len is in ws
__global__ void llg_quat_apply_2(
//	const int nx, const int ny, const int offset,
	const int nxyz,
	double* ssx, double* ssy, double* ssz, // src
	double* wx, double* wy, double* wz, double* ww,
	const double dt, double alpha, const double* d_alpha, double gamma, const double* d_gamma)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i >= nxyz)
		return;

	if(ww[i] == 0) // dst = src  in _3
	{
		return;
	}

// dS    -g           a
// -- = ---- S X (H +---S X H)
// dt   1+aa         |S|
	double4 qVec;
	qVec.x = ssx[i]; 
	qVec.y = ssy[i];
	qVec.z = ssz[i];
	qVec.w = 0;
			
	// the 0.5 is for the quaternions
	double gadt;
	if(d_alpha)
	{
		if(d_gamma)
		{
			gadt = (0.5 * d_gamma[i] * dt) / (1.0 + d_alpha[i] * d_alpha[i]);
		}
		else
		{
			gadt = (0.5 * gamma * dt) / (1.0 + d_alpha[i] * d_alpha[i]);
		}
	}
	else
	{
		if(d_gamma)
		{
			gadt = (0.5 * d_gamma[i] * dt) / (1.0 + alpha * alpha);
		}
		else
		{
			gadt = (0.5 * gamma * dt) / (1.0 + alpha * alpha);
		}
	}
	
	const double theta = ww[i] * gadt;

	double cost, sint;
	sincos(theta, &sint, &cost);
	const double ihlen = FAST_DIV(1, ww[i]);

	double4 qRot;
	qRot.x = sint * wx[i] * ihlen; 
	qRot.y = sint * wy[i] * ihlen;
	qRot.z = sint * wz[i] * ihlen;
	qRot.w = cost;

	//this is the rotation: qRes = qRot qVec qRot*
	double4 qRes = qmultXYZ(qmult(qRot, qVec), qconjugate(qRot));

	wx[i] = qRes.x;
	wy[i] = qRes.y;
	wz[i] = qRes.z;
	
	// (wx, wy, wz) = (qRot qVec) conj(qRot)
	// ww = unimportant
}

// _3 normalize
__global__ void llg_quat_apply_3(
//	const int nx, const int ny, const int offset,
	const int nxyz,
	double*  wx, double*  wy, double*  wz, double* ww,
	double* ssx, double* ssy, double* ssz, double* sms,
 	double* dsx, double* dsy, double* dsz, double* dms)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i >= nxyz)
		return;

	dms[i] = sms[i];
	
	if(ww[i] == 0)
	{
		dsx[i] = ssx[i];
		dsy[i] = ssy[i];
		dsz[i] = ssz[i];
		return;
	}
	
	//using ww as temp var. saves a reg?
	ww[i] = sqrt(wx[i]*wx[i] + wy[i]*wy[i] + wz[i]*wz[i]);
	
	if(ww[i] == 0)
	{
		dsx[i] = ssx[i];
		dsy[i] = ssy[i];
		dsz[i] = ssz[i];
		return;
	}
	
	
	ww[i] = sms[i] / ww[i];

	// (wx, wy, wz) = (qRot qVec) conj(qRot)
	// ww = | S | / |(wx, wy, wz)|
}

// _4
__global__ void llg_quat_apply_4(
//	const int nx, const int ny, const int offset,
	const int nxyz,
	double*  wx, double*  wy, double*  wz, double* ww,
 	double* dsx, double* dsy, double* dsz)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i >= nxyz)
		return;

	if(ww[i] == 0)
	{
		return;
	}
	
	dsx[i] = wx[i] * ww[i];
	dsy[i] = wy[i] * ww[i];
	dsz[i] = wz[i] * ww[i];
}

// 	cuda_llg_quat_apply(
// 			    dmdt->d_x,     dmdt->d_y,     dmdt->d_z,     dmdt->d_ms,
//             dmdt->d_hx[T], dmdt->d_hy[T], dmdt->d_hz[T],
// 			dmdt->d_hx[S], dmdt->d_hy[S], dmdt->d_hz[S],
// 			          d_ws1,         d_ws2,         d_ws3,         d_ws4,
// 			alpha, dt, gamma);	

	
void cuda_llg_quat_apply(const int nx, const int ny, const int nz,
	double* dsx, double* dsy, double* dsz, double* dms, //dest (spinto)
	double* ssx, double* ssy, double* ssz, double* sms, // src (spinfrom)
	double* ddx, double* ddy, double* ddz, double* dds, // dm/dt spins
    double* htx, double* hty, double* htz,              // dm/dt thermal fields
	double* dhx, double* dhy, double* dhz,              // dm/dt fields
	double* ws1, double* ws2, double* ws3, double* ws4,
	const double dt, const double alpha, const double* d_alpha, const double gamma, const double* d_gamma,
	int thermalOnlyFirstTerm)
{
	const int nxyz = nx*ny*nz;
	const int threads = 512;
	const int blocks = nxyz / threads + 1;

	// _1 calculates rhs of S x (damped field)
	// the (damped field) is done with dm/dt terms
	// result is stored in W
	//	const int nx, const int ny, const int offset,
	if(thermalOnlyFirstTerm)
		llg_quat_apply_1<1><<<blocks, threads>>>(nxyz,
							alpha, d_alpha,
							ddx, ddy, ddz, dds,
							dhx, dhy, dhz,
							htx, hty, htz,
							ws1, ws2, ws3, ws4);
	else
		llg_quat_apply_1<0><<<blocks, threads>>>(nxyz,
							alpha, d_alpha,
							ddx, ddy, ddz, dds,
							dhx, dhy, dhz,
							htx, hty, htz,
							ws1, ws2, ws3, ws4);
	CHECK
	
	// spinfrom x W (via quats)
	llg_quat_apply_2<<<blocks, threads>>>(nxyz,
				ssx, ssy, ssz,
				ws1, ws2, ws3, ws4,
				dt, alpha, d_alpha, gamma, d_gamma);
	CHECK

	// normalize
	llg_quat_apply_3<<<blocks, threads>>>(nxyz,
					ws1, ws2, ws3, ws4,
					ssx, ssy, ssz, sms,
					dsx, dsy, dsz, dms);
	CHECK
	
	// store in (spinto)
	llg_quat_apply_4<<<blocks, threads>>>(nxyz,
					ws1, ws2, ws3, ws4,
					dsx, dsy, dsz);
	CHECK
}

