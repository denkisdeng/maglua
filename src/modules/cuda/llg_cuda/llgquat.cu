#include <hip/hip_runtime.h>

#include <math_functions.h>
#include <stdio.h>

#define CROSS(v, a, b) \
	v.x = a.y * b.z - a.z * b.y; \
	v.y = a.z * b.x - a.x * b.z; \
	v.z = a.x * b.y - a.y * b.x;
	
#if __CUDA_ARCH__ >= 200
#define FAST_DIV(x,y) __ddiv_rn(x,y)
#else
#define FAST_DIV(x,y) ((x)/(y))
#endif

#define CHECK \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  __FILE__, __LINE__-1, hipGetErrorString(i));\
}


__device__ double4 qconjugate(const double4 q)
{
	double4 r;
	r.x = -1.0 * q.x;
	r.y = -1.0 * q.y;
	r.z = -1.0 * q.z;
	r.w =        q.w;
	return r;
}

__device__ double4 qmult(const double4 a, const double4 b)
{
	double4 ab;

	ab.w = a.w*b.w - a.x*b.x - a.y*b.y - a.z*b.z;
	ab.x = a.w*b.x + b.w*a.x  + a.y*b.z - a.z*b.y;
	ab.y = a.w*b.y + b.w*a.y  + a.z*b.x - a.x*b.z;
	ab.z = a.w*b.z + b.w*a.z  + a.x*b.y - a.y*b.x;

	return ab;
}

// quat mult without calculating the W component
__device__ double4 qmultXYZ(const double4 a, const double4 b)
{
	double4 ab;

	ab.x = a.w*b.x + b.w*a.x  + a.y*b.z - a.z*b.y;
	ab.y = a.w*b.y + b.w*a.y  + a.z*b.x - a.x*b.z;
	ab.z = a.w*b.z + b.w*a.z  + a.x*b.y - a.y*b.x;
	ab.w = 0;

	return ab;
}

// _1 will compute rhs and store in ws
// dS    -g           a
// -- = ---- S X (H +---S X H)
// dt   1+aa         |S|
__global__ void llg_quat_apply_1(
//	const int nx, const int ny, const int offset,
	const int nxyz,
	double alpha,
	double* sx, double* sy, double* sz, double* sms,
	double* hx, double* hy, double* hz,
	// ws1, ws2, ws3,     ws4
	double* wx, double* wy, double* wz, double* ww) 
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i >= nxyz)
		return;
	
	wx[i] = sy[i]*hz[i] - sz[i]*hy[i];
	wy[i] = sz[i]*hx[i] - sx[i]*hz[i];
	wz[i] = sx[i]*hy[i] - sy[i]*hx[i];
	
	ww[i] = sms[i];
	if(ww[i] != 0)
	{
		ww[i] = FAST_DIV(alpha, ww[i]);
	}
	wx[i] *= ww[i];
	wy[i] *= ww[i];
	wz[i] *= ww[i];
	
	wx[i] += hx[i];
	wy[i] += hy[i];
	wz[i] += hz[i];

	ww[i] = sqrt(wx[i]*wx[i] + wy[i]*wy[i] + wz[i]*wz[i]);
	
	//(wx, wy, wz) = (a / |S|) S x H
	// ww = | (wx, wy, wz) |
}

// _2 will compute the rest
// dS    -g           a
// -- = ---- S X (H +---S X H)
// dt   1+aa         |S|
// the rhs vec and len is in ws
__global__ void llg_quat_apply_2(
//	const int nx, const int ny, const int offset,
	const int nxyz,
	double* ssx, double* ssy, double* ssz, // src
	double* wx, double* wy, double* wz, double* ww,
	double alpha, double gadt)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i >= nxyz)
		return;

	if(ww[i] == 0) // dst = src  in _3
	{
		return;
	}

// dS    -g           a
// -- = ---- S X (H +---S X H)
// dt   1+aa         |S|
	double4 qVec;
	qVec.x = ssx[i]; 
	qVec.y = ssy[i];
	qVec.z = ssz[i];
	qVec.w = 0;
			
	const double theta = ww[i] * gadt;

	double cost, sint;
	sincos(theta, &sint, &cost);
	const double ihlen = FAST_DIV(1, ww[i]);

	double4 qRot;
	qRot.x = sint * wx[i] * ihlen; 
	qRot.y = sint * wy[i] * ihlen;
	qRot.z = sint * wz[i] * ihlen;
	qRot.w = cost;

	//this is the rotation: qRes = qRot qVec qRot*
	double4 qRes = qmultXYZ(qmult(qRot, qVec), qconjugate(qRot));

	wx[i] = qRes.x;
	wy[i] = qRes.y;
	wz[i] = qRes.z;
	
	// (wx, wy, wz) = (qRot qVec) conj(qRot)
	// ww = unimportant
}

// _3 normalize
__global__ void llg_quat_apply_3(
//	const int nx, const int ny, const int offset,
	const int nxyz,
	double*  wx, double*  wy, double*  wz, double* ww,
	double* ssx, double* ssy, double* ssz, double* sms,
 	double* dsx, double* dsy, double* dsz, double* dms)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i >= nxyz)
		return;

	dms[i] = sms[i];
	
	if(ww[i] == 0)
	{
		dsx[i] = ssx[i];
		dsy[i] = ssy[i];
		dsz[i] = ssz[i];
		return;
	}
	
	//using ww as temp var. saves a reg?
	ww[i] = sqrt(wx[i]*wx[i] + wy[i]*wy[i] + wz[i]*wz[i]);
	
	if(ww[i] == 0)
	{
		dsx[i] = ssx[i];
		dsy[i] = ssy[i];
		dsz[i] = ssz[i];
		return;
	}
	
	
	ww[i] = sms[i] / ww[i];

	// (wx, wy, wz) = (qRot qVec) conj(qRot)
	// ww = | S | / |(wx, wy, wz)|
}

// _4
__global__ void llg_quat_apply_4(
//	const int nx, const int ny, const int offset,
	const int nxyz,
	double*  wx, double*  wy, double*  wz, double* ww,
 	double* dsx, double* dsy, double* dsz)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i >= nxyz)
		return;

	if(ww[i] == 0)
	{
		return;
	}
	
	dsx[i] = wx[i] * ww[i];
	dsy[i] = wy[i] * ww[i];
	dsz[i] = wz[i] * ww[i];
}



void cuda_llg_quat_apply(const int nx, const int ny, const int nz,
	double* dsx, double* dsy, double* dsz, double* dms, //dest
	double* ssx, double* ssy, double* ssz, double* sms, // src
	double* hx, double* hy, double* hz,
	double* ws1, double* ws2, double* ws3, double* ws4,
	const double alpha, const double dt, const double gamma)
{
	// the 0.5 is for the quaternions
	double gadt = (0.5 * gamma * dt) / (1.0 + alpha * alpha);

	const int nxyz = nx*ny*nz;
	const int threads = 512;
	const int blocks = nxyz / threads + 1;

	llg_quat_apply_1<<<blocks, threads>>>(nxyz,
					alpha,
					ssx, ssy, ssz, sms,
						hx,  hy,  hz,
					ws1, ws2, ws3, ws4);
	CHECK
	
	llg_quat_apply_2<<<blocks, threads>>>(nxyz,
					ssx, ssy, ssz,
					ws1, ws2, ws3, ws4,
					alpha, gadt);
	CHECK

	llg_quat_apply_3<<<blocks, threads>>>(nxyz,
					ws1, ws2, ws3, ws4,
					ssx, ssy, ssz, sms,
					dsx, dsy, dsz, dms);
	CHECK
	
	llg_quat_apply_4<<<blocks, threads>>>(nxyz,
					ws1, ws2, ws3, ws4,
					dsx, dsy, dsz);
	CHECK
}
