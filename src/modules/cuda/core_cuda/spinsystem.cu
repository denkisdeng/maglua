#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_complex.h>

#include "spinsystem.hpp"
#include <stdio.h>

#define KCHECK \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  __FILE__, __LINE__-1, hipGetErrorString(i));\
}

#define KCHECK_FL(f,l) \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  f, l, hipGetErrorString(i));\
}


#define SEGFAULT \
{ \
	long* i = 0; \
	*i = 5; \
}

#define CHECKCALL_FL(expression, file, line) \
{ \
	const hipError_t err = (expression); \
	if(err != hipSuccess) \
	{ \
		printf("(%s:%i) %s => (%i)%s\n", file, line, #expression, err, hipGetErrorString(err)); \
	} \
}

#define CHECKCALL(expression)  CHECKCALL_FL(expression, __FILE__, __LINE__)


void ss_d_make3DArray(double** v, int nx, int ny, int nz)
{
	CHECKCALL(hipMalloc(v, sizeof(double) * nx * ny * nz));
}

void ss_d_free3DArray(double* v)
{
	CHECKCALL(hipFree(v));
}

void ss_h_make3DArray(double** v, int nx, int ny, int nz)
{
	CHECKCALL(hipHostMalloc(v, sizeof(double) * nx * ny * nz));
}

void ss_h_free3DArray(double* v)
{
	CHECKCALL(hipHostFree(v));
}

void ss_copyDeviceToHost_(double* dest, double* src, int nxyz, const char* file, const unsigned int line)
{
	CHECKCALL_FL(hipMemcpy(dest, src, sizeof(double)*nxyz, hipMemcpyDeviceToHost), file, line);
}

void ss_copyHostToDevice_(double* dest, double* src, int nxyz, const char* file, const unsigned int line)
{
// 	printf("%p <- %p (%i)\n", dest, src, nxyz);
	CHECKCALL_FL(hipMemcpy(dest, src, sizeof(double)*nxyz, hipMemcpyHostToDevice), file, line);
}





__global__ void addValue(double* dest, const int n, double* s1, double* s2)
{
	const int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(idx >= n)
		return;
	dest[idx] = s1[idx] + s2[idx];
}

void ss_d_add3DArray(double* d_dest, int nx, int ny, int nz, double* d_src1, double* d_src2)
{
	const int threads = 256;
	const int nxyz = nx*ny*nz;
	const int blocks = nxyz / threads + 1;

	addValue<<<blocks, threads>>>(d_dest, nxyz, d_src1, d_src2);
	KCHECK
}


__global__ void setArray(
	double* dest, const int nxyz, double value)
{
	const int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(idx >= nxyz)
		return;
	
	dest[idx] = value;
}

void ss_d_set3DArray_(double* d_v, int nx, int ny, int nz, double value, const char* file, const unsigned int line)
{
	const int threads = 256;
	const int nxyz = nx*ny*nz;
	const int blocks = nxyz / threads + 1;
	
	setArray<<<blocks, threads>>>(d_v, nxyz, value);
	KCHECK_FL(file, line)
}



__global__ void absDiffArrays(
	double* dest, double* src1, double* src2, const int nxyz)
{
	const int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(idx >= nxyz)
		return;
	
	dest[idx] = fabs(src1[idx] - src2[idx]);
}

void ss_d_absDiffArrays_(double* d_dest, double* d_src1, double* d_src2, int nxyz, const char* file, const unsigned int line)
{
	const int threads = 256;
	const int blocks = nxyz / threads + 1;
	
	absDiffArrays<<<blocks, threads>>>(d_dest, d_src1, d_src2, nxyz);
	KCHECK_FL(file, line)
}



template <unsigned int blockSize>
__device__ void warpReduce(volatile double *sdata, unsigned int tid)
{
	if (blockSize >=  64) sdata[tid] += sdata[tid + 32];
	if (blockSize >=  32) sdata[tid] += sdata[tid + 16];
	if (blockSize >=  16) sdata[tid] += sdata[tid +  8];
	if (blockSize >=   8) sdata[tid] += sdata[tid +  4];
	if (blockSize >=   4) sdata[tid] += sdata[tid +  2];
	if (blockSize >=   2) sdata[tid] += sdata[tid +  1];
}

template <unsigned int blockSize>
__global__ void reduce_sum_kernel(double *g_odata, double *g_idata, unsigned int n)
{
// 	extern __shared__ double sdata[];
	__shared__ double sdata[blockSize];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*2*gridDim.x;
	sdata[tid] = 0;
	while (i < n)
	{
		sdata[tid] += g_idata[i] + g_idata[i+blockSize];
		i += gridSize;
	}

	__syncthreads();
	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			sdata[tid] += sdata[tid + 256];
		}
	__syncthreads();
	}
	
	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			sdata[tid] += sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128)
	{
		if (tid <   64)
		{
			sdata[tid] += sdata[tid +   64];
		}
		__syncthreads();
	}

	if(tid < 32)
		warpReduce<blockSize>(sdata, tid);
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

#define BS 64
double ss_reduce3DArray_sum(double* d_v, double* d_ws1, double* h_ws1, int nx, int ny, int nz)
{
	int blocks = (nx*ny*nz) / BS + 1;
	
	unsigned int n = nx*ny*nz;
	
	reduce_sum_kernel<BS><<<blocks, BS>>>(d_ws1, d_v, n);
	KCHECK

	CHECKCALL(hipMemcpy(h_ws1, d_ws1, sizeof(double)*blocks, hipMemcpyDeviceToHost));
	
	
	for(int i=1; i<blocks; i++)
		h_ws1[0] += h_ws1[i];
	return h_ws1[0];
}



void ss_d_copyArray(double* d_dest, double* d_src, int nxyz)
{
	hipMemcpy(d_dest, d_src, sizeof(double)*nxyz, hipMemcpyDeviceToDevice);
}


