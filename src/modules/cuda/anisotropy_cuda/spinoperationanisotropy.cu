#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_complex.h>

#include "spinoperationanisotropy.hpp"


__global__ void do_anisotropy_N(const double global_scale,
	const double** d_sx_N, const double** d_sy_N, const double** d_sz_N,
	const double* d_nx, const double* d_ny, const double* d_nz, const double* d_k,
	double** d_hx_N, double** d_hy_N, double** d_hz_N, 
	const int nxyz, const int n)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= nxyz) return;
	
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if(j >= n) return;
	
	const double* d_sx = d_sx_N[j];
	const double* d_sy = d_sy_N[j];
	const double* d_sz = d_sz_N[j];

	double* d_hx = d_hx_N[j];
	double* d_hy = d_hy_N[j];
	double* d_hz = d_hz_N[j];
	
	const double ms2 = d_sx[i]*d_sx[i] + d_sy[i]*d_sy[i] + d_sz[i]*d_sz[i];
	
	if(ms2 > 0)
	{
		const double SpinDotEasyAxis = 
			d_sx[i]*d_nx[i] + d_sy[i]*d_ny[i] + d_sz[i]*d_nz[i];
	
		const double v = global_scale * 2.0 * d_k[i] * SpinDotEasyAxis / ms2;
		
		d_hx[i] = d_nx[i] * v;
		d_hy[i] = d_ny[i] * v;
		d_hz[i] = d_nz[i] * v;
	}
	else
	{
		d_hx[i] = 0;
		d_hy[i] = 0;
		d_hz[i] = 0;
	}
}

void cuda_anisotropy_N(const double global_scale,
	const double** d_sx, const double** d_sy, const double** d_sz,
	const double* d_nx, const double* d_ny, const double* d_nz, const double* d_k,
	double** d_hx, double** d_hy, double** d_hz,
	const int nxyz, const int n)
{
	const int threadsX = 256;
	const int blocksX = nxyz / threadsX + 1;

	const int threadsY = 1;
	const int blocksY = n;
	
	dim3 gd(blocksX, blocksY);
	dim3 bd(threadsX, threadsY);
	
	
// 	const int blocksx = nx / 32 + 1;
// 	const int blocksy = ny / 32 + 1;

// 	dim3 blocks(blocksx, blocksy);
// 	dim3 threads(32, 32);

// 	for(int z=0; z<nz; z++)
// 	{
// 		const int offset = z * nx * ny;
		do_anisotropy_N<<<gd, bd>>>(
				global_scale,
				d_sx, d_sy, d_sz,
				d_nx, d_ny, d_nz, d_k,
				d_hx, d_hy, d_hz, 
				nxyz, n);
// 	}
}





__global__ void do_anisotropy_compressed_N(const double global_scale,
	const double** d_sx_N, const double** d_sy_N, const double** d_sz_N,
	const double* d_LUT, const char* d_idx,
	double** d_hx_N, double** d_hy_N, double** d_hz_N, 
	const int nxyz, int n)
{
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= nxyz) return;
	
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if(j >= n) return;
	
	const double* d_sx = d_sx_N[j];
	const double* d_sy = d_sy_N[j];
	const double* d_sz = d_sz_N[j];

	double* d_hx = d_hx_N[j];
	double* d_hy = d_hy_N[j];
	double* d_hz = d_hz_N[j];
	
	
	
	const double nx = d_LUT[d_idx[i]*4+0];
	const double ny = d_LUT[d_idx[i]*4+1];
	const double nz = d_LUT[d_idx[i]*4+2];
	const double k  = d_LUT[d_idx[i]*4+3];
	
	const double ms2 = d_sx[i]*d_sx[i] + d_sy[i]*d_sy[i] + d_sz[i]*d_sz[i];
	
	if(ms2 > 0)
	{
		const double SpinDotEasyAxis = 
			d_sx[i]*nx + d_sy[i]*ny + d_sz[i]*nz;
	
		const double v = global_scale * 2.0 * k * SpinDotEasyAxis / ms2;
		
		d_hx[i] = nx * v;
		d_hy[i] = ny * v;
		d_hz[i] = nz * v;
	}
	else
	{
		d_hx[i] = 0;
		d_hy[i] = 0;
		d_hz[i] = 0;
	}
}


void cuda_anisotropy_compressed_N(const double global_scale,
	const double** d_sx, const double** d_sy, const double** d_sz,
	const double* d_LUT, const char* d_idx,
	double** d_hx, double** d_hy, double** d_hz,
	const int nxyz, const int n)
{
	const int threadsX = 256;
	const int blocksX = nxyz / threadsX + 1;

	const int threadsY = 1;
	const int blocksY = n;
	
	dim3 gd(blocksX, blocksY);
	dim3 bd(threadsX, threadsY);
	
	
// 	const int blocksx = nx / 32 + 1;
// 	const int blocksy = ny / 32 + 1;

// 	dim3 blocks(blocksx, blocksy);
// 	dim3 threads(32, 32);

// 	for(int z=0; z<nz; z++)
// 	{
// 		const int offset = z * nx * ny;
		do_anisotropy_compressed_N<<<gd, bd>>>(
				global_scale,
				d_sx, d_sy, d_sz,
				d_LUT, d_idx,
				d_hx, d_hy, d_hz, 
				nxyz, n);
		
		
// 	const int blocks = nxyz / 1024 + 1;
// 	const int threads = 1024;
// 
// 	do_anisotropy_compressed<<<blocks, threads>>>(
// 		    global_scale,
// 			d_sx, d_sy, d_sz,
// 			d_LUT, d_idx,
// 			d_hx, d_hy, d_hz, 
// 			nxyz);
}
