#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>

#define CHECK \
{ \
	const hipError_t i = hipGetLastError();\
	if(i) \
		printf("(%s:%i) %s\n",  __FILE__, __LINE__-1, hipGetErrorString(i));\
}

#define IDX_PATT(a, b) \
	const int a = blockDim.x * blockIdx.x + threadIdx.x; \
	const int b = blockDim.y * blockIdx.y + threadIdx.y;
	
	
// FOOBAR = (2.0 * alpha * temperature) / (dt * gamma)
template <unsigned int twiddle>
__global__ void do_thermal32(	
		const float* d_rng6,  
		float FOOBAR, float* d_scale,
		float* d_hx, float* d_hy, float* d_hz, float* d_ms,
		const int nx, const int ny, const int offset)
{
	IDX_PATT(x, y);
	if(x >= nx || y >= ny)
		return;
	const int idx = x + y*nx + offset;
	
	const float ms = d_ms[idx];
	if(ms != 0)
	{
		const float stddev = sqrt((FOOBAR * d_scale[idx]) / ms);
		d_hx[idx] = stddev * d_rng6[idx*6+0+twiddle*3];
		d_hy[idx] = stddev * d_rng6[idx*6+1+twiddle*3];
		d_hz[idx] = stddev * d_rng6[idx*6+2+twiddle*3];
	}
	else
	{
		d_hx[idx] = 0;
		d_hy[idx] = 0;
		d_hz[idx] = 0;
	}
}


void cuda_thermal32(const float* d_rng6, const int twiddle, 
	float alpha, float gamma, float dt, float temperature,
	float* d_hx, float* d_hy, float* d_hz, float* d_ms,
	float* d_scale,
	const int nx, const int ny, const int nz)
{
	const float FOOBAR =  (2.0 * alpha * temperature) / (dt * gamma);
	const int _blocksx = nx / 32 + 1;
	const int _blocksy = ny / 32 + 1;
	dim3 blocks(_blocksx, _blocksy);
	dim3 threads(32,32);
	
	if(twiddle == 0)
	{
		for(int i=0; i<nz; i++)
		{
			do_thermal32<0><<<blocks, threads>>>(d_rng6, FOOBAR, d_scale, d_hx, d_hy, d_hz, d_ms, nx, ny, nx*ny*i);
			CHECK
		}
	}
	else
	{
		for(int i=0; i<nz; i++)
		{
			do_thermal32<1><<<blocks, threads>>>(d_rng6, FOOBAR, d_scale, d_hx, d_hy, d_hz, d_ms, nx, ny, nx*ny*i);
			CHECK
		}
	}
}
